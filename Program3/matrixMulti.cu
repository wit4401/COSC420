/*
 Assuming each matrix is nxn we can construct a program using
 CUDA to implement matrix multiplication
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 3*3 /* Size of arrays (Size of matrix: "sqrt(SIZE) x sqrt(SIZE)" */
#define GRID 1 /* Size of the grid (i.e. dimensions "Grid x Grid x Grid") */

__global__ void matrix_multiply(int *a,int *b, int *res, int width){
    int row = threadIdx.y+(width*blockIdx.y);
    int col = threadIdx.x+(width*blockIdx.x);
    int sum = 0;

    if((row<SIZE) && (col<SIZE)){
        for(int i=0;i<width;i++)
            sum+=a[(row*width)+i]*b[(i*width)+col];
        res[(row*width)+col]=sum;
    }
}

/* Prints out the given matrix */
void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)-1){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    int len=sqrt(SIZE); /* length of each row/column */
    srand(time(NULL));

    //allocate appropriate memory to each dynamic array
    hipMallocManaged(&matA,sizeof(int)*SIZE);
    hipMallocManaged(&matB,sizeof(int)*SIZE);
    hipMallocManaged(&result,sizeof(int)*SIZE);

    // assigns random number to each index of the dynamic arrays to be multiplied
    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%11;
        matB[i]=rand()%11;
    }

    dim3 grid_size(GRID); //stores our grid dimensions
    dim3 block_size(SIZE/GRID);//stores our block dimensions

    //call the kernel with the appropriate grid and block dimensions
    matrix_multiply<<<grid_size,block_size>>>(matA,matB,result,len);
    hipDeviceSynchronize();

    //print out results from computation
    printMatrix(matA,len);
    puts("\nTimes");
    printMatrix(matB,len);
    puts("\nEquals");
    printMatrix(result,len);
    puts("");


    //clean up memory
    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    return 0;
}
