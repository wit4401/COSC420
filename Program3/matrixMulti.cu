#include "hip/hip_runtime.h"
/*
 Assuming each matrix is nxn we can construct a program using
 CUDA to implement matrix multiplication
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#define SIZE 1024

__global__ void matrix_multiply(int *a,int *b, int *res){

}

int main(int argc, char **argv){
    int matA*,matB*,result;

    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);

    matrix_multiply<<<1,1>>>(matA,matB,result);
    

    return 0;
}