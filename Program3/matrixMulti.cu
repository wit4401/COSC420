/*
 Assuming each matrix is nxn we can construct a program using
 CUDA to implement matrix multiplication
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 10000
#define GRID 10

__global__ void matrix_multiply(int *a,int *b, int *res, int n){

}

/* Prints out the given matrix */
void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)-1){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    int len=sqrt(SIZE);/* length of each row/column */
    srand(time(NULL));

    //allocate appropriate memory to each dynamic array
    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);
    hipMalloc(&result,sizeof(int)*SIZE);

    // assigns random number to each index of the dynamic arrays to be added
    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%101;
        matB[i]=rand()%101;
    }

    dim3 grid_size(GRID); //stores our grid dimensions
    dim3 block_size(SIZE/GRID);//stores our block dimensions

    //call the kernel with the appropriate grid and block dimensions
    matrix_multiply<<<grid_size,block_size>>>(matA,matB,result,SIZE);
    hipDeviceSynchronize();

    //print out results from computation
    printMatrix(matA,len);
    puts("\nPlus");
    printMatrix(matB,len);
    puts("\nEquals");
    printMatrix(result,len);
    puts("");

    //clean up memory
    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    return 0;
}