/*
 Assuming each matrix is nxn we can construct a program using
 CUDA to implement matrix multiplication
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 10000 /* Size of arrays (Size of matrix: "sqrt(SIZE) x sqrt(SIZE)" */
#define GRID 10 /* Size of the grid (i.e. dimensions "Grid x Grid x Grid") */

__global__ void matrix_multiply(int *a,int *b, int *res, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    if(row<n&&col<n){
        int temp = 0;
        for(int i=0;i<n;i++)
            temp+=a[row*n+i]*b[i*n+col];
        res[row*n+col]=temp;
    }
}

/* Prints out the given matrix */
void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)-1){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    int len=sqrt(SIZE);/* length of each row/column */
    srand(time(NULL));

    //allocate appropriate memory to each dynamic array
    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);
    hipMalloc(&result,sizeof(int)*SIZE);

    // assigns random number to each index of the dynamic arrays to be added
    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%11;
        matB[i]=rand()%11;
    }

    dim3 grid_size(GRID); //stores our grid dimensions
    dim3 block_size(SIZE/GRID);//stores our block dimensions

    //call the kernel with the appropriate grid and block dimensions
    matrix_multiply<<<grid_size,block_size>>>(matA,matB,result,SIZE);
    hipDeviceSynchronize();

    //print out results from computation
    printMatrix(matA,len);
    puts("\nPlus");
    printMatrix(matB,len);
    puts("\nEquals");
    printMatrix(result,len);
    puts("");

    //clean up memory
    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    return 0;
}