#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 10000 /* Size of arrays (Size of matrix: "sqrt(SIZE) x sqrt(SIZE)" */
#define GRID 10 /* Size of the grid (i.e. dimensions "Grid x Grid x Grid") */

/* Our kernel, i.e. the instructions for each thread in this case adding together numbers in the same spot */
__global__ void matrix_addition(int *a,int *b, int *res,int n){
    int i = threadIdx.x+(blockDim.x*blockIdx.x);
    if (i<n)
        res[i] = a[i]+b[i];
}

/* Prints out the given matrix */
void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)-1){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    int len=sqrt(SIZE);/* length of each row/column */
    srand(time(NULL));
    
    //allocate appropriate memory to each dynamic array
    hipMallocManaged(&matA,SIZE*sizeof(int));
    hipMallocManaged(&matB,SIZE*sizeof(int));
    hipMallocManaged(&result,SIZE*sizeof(int));

    // assigns random number to each index of the dynamic arrays to be added
    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%101;
        matB[i]=rand()%101;
    }

    dim3 grid_size(GRID); //stores our grid dimensions
    dim3 block_size(SIZE/GRID);//stores our block dimensions

    //call the kernel with the appropriate grid and block dimensions
    matrix_addition<<<grid_size,block_size>>>(matA,matB,result,SIZE);
    hipDeviceSynchronize();

    //print out results from computation
    printMatrix(matA,len);
    puts("\nPlus");
    printMatrix(matB,len);
    puts("\nEquals");
    printMatrix(result,len);
    puts("");

    //clean up memory
    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    exit(0);
}
