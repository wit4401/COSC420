#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 10000
#define GRID 10

__global__ void matrix_addition(int *a,int *b, int *res,int n){
    int i = threadIdx.x+(blockDim.x*blockIdx.x);
    if (i<n)
        res[i] = a[i]+b[i];
}

void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)-1){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    int len=sqrt(SIZE);
    srand(time(NULL));

    hipMallocManaged(&matA,SIZE*sizeof(int));
    hipMallocManaged(&matB,SIZE*sizeof(int));
    hipMallocManaged(&result,SIZE*sizeof(int));

    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%101;
        matB[i]=rand()%101;
    }

    dim3 grid_size(GRID);
    dim3 block_size(SIZE/GRID);

    matrix_addition<<<grid_size,block_size>>>(matA,matB,result,SIZE);
    hipDeviceSynchronize();

    printMatrix(matA,len);
    puts("\nPlus");
    printMatrix(matB,len);
    puts("\nEquals");
    printMatrix(result,len);
    puts("");

    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    exit(0);
}
