#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#define SIZE 25

__global__ void matrix_addition(int *a,int *b, int *res,int n){
    int i = threadIdx.x;
    if (i<n)
        res[i] = a[i]+b[i];
}

void printMatrix(int *mat, int len_row){
    int multiple=1;
    for (int i=0;i<SIZE;i++){
        if(i == (multiple*len_row)){
            printf("%d\n",mat[i]);
            multiple++;
        }
        else
            printf("%d ",mat[i]);
    }
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    srand(time(NULL));

    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);
    hipMalloc(&result,sizeof(int)*SIZE);

    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%101;
        matB[i]=rand()%101;
        result[i]=0;
    }

    matrix_addition<<<1,1>>>(matA,matB,result,SIZE);
    hipDeviceSynchronize();

    len=sqrt(SIZE);
    printMatrix(matA,SIZE,len);
    puts("Plus");
    printMatrix(matB,SIZE,len);
    puts("Equals");
    printMatrix(result,SIZE,len);

    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    exit(0);
}