#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#define SIZE 2500

__global__ void matrix_addition(int *a,int *b, int *res,int n){
    int i = threadIdx.x;
    if (i<n)
        res = a[i]+b[i]
    
}

int main(int argc, char **argv){
    int *matA,*matB,*result;
    srand(time(NULL));

    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);
    hipMalloc(&result,sizeof(int)*SIZE);

    for(int i=0;i<SIZE;i++){
        matA[i]=rand()%101;
        matB[i]=rand()%101;
        result[i]=0;
    }

    matrix_addition<<<1,1>>>(matA,matB,result);

    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    return 0;
}