#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define SIZE 1024

__global__ void matrix_addition(int *a,int *b, int *res){

}

int main(int argc, char **argv){
    int *matA,*matB,*result;

    hipMalloc(&matA,sizeof(int)*SIZE);
    hipMalloc(&matB,sizeof(int)*SIZE);
    hipMalloc(&result,sizeof(int)*SIZE);

    matrix_addition<<<1,1>>>(matA,matB,result);

    hipFree(matA);
    hipFree(matB);
    hipFree(result);
    
    return 0;
}